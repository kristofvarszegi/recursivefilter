#include "hip/hip_runtime.h"
#include "Logger.hpp"
#include "gpufuncs.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <stdexcept>

#define SHMEM_PAD_X 1
#define PRINT_LIMIT_X 8
#define PRINT_LIMIT_Y 8
#define SAVE_TABLES_TO_CSV false

namespace gpuacademy {

namespace config {

size_t get_blockdim_2dgrid_x(config::BLOCK_SIZE_CLASS bsc) {
  switch (bsc) {
  case config::BLOCK_SIZE_CLASS::SMALL:
    return 4;
    break;
  case config::BLOCK_SIZE_CLASS::LARGE:
    return 28;
    break;
  default:
    throw std::runtime_error("Invalid block size class: " +
                             std::to_string(bsc));
  }
}

size_t get_blockdim_2dgrid_y(config::BLOCK_SIZE_CLASS bsc) {
  switch (bsc) {
  case config::BLOCK_SIZE_CLASS::SMALL:
    return 4;
    break;
  case config::BLOCK_SIZE_CLASS::LARGE:
    return 28;
    break;
  default:
    throw std::runtime_error("Invalid block size class: " +
                             std::to_string(bsc));
  }
}

size_t get_blockdim_1dgrid_x(config::BLOCK_SIZE_CLASS bsc) {
  switch (bsc) {
  case config::BLOCK_SIZE_CLASS::SMALL:
    return 7;
    break;
  case config::BLOCK_SIZE_CLASS::LARGE:
    return 64;
    break;
  default:
    throw std::runtime_error("Invalid block size class: " +
                             std::to_string(bsc));
  }
}

} // namespace config

inline void chk_cu_err(hipError_t code) {
  if (code != hipSuccess) {
    const char *buf;
    hipDrvGetErrorString(code, &buf);
    throw std::runtime_error("Error: " + std::string(buf) + "\n");
  }
}

inline void chk_cu_err(hipError_t code) {
  if (code != hipSuccess) {
    throw std::runtime_error("Error: " + std::string(hipGetErrorString(code)) +
                             "\n");
  }
}

__global__ void recursivefilter_step1_inblocksdownright(
    const float* __restrict__ input, int num_rows, int num_cols, float filter_coeff_0,
    float filter_coeff_1, float* __restrict__ blockwise_colwise_sums,
    float* __restrict__ blockwise_rowwise_sums) {
  const int global_tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_tid_y = blockIdx.y * blockDim.x + threadIdx.x;
  // Yes, threadIdx.x (not .y), as we have a 1D thread array within a thread
  // block
  extern __shared__ float colwisesums_thisblock[];

  if (global_tid_x < num_cols) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
#pragma unroll
    for (int y_in_thisblock = 0; y_in_thisblock < blockDim.x;
         ++y_in_thisblock) {
      if (blockIdx.y * blockDim.x + y_in_thisblock < num_rows) {
        aggregated_sum =
            filter_coeff_0 *
            __ldg(
                (const float *)&input[global_tid_x + (blockIdx.y * blockDim.x +
                                                      y_in_thisblock) *
                                                         num_cols]);
        aggregated_sum += filter_coeff_1 * prev_aggregated_sum;
      }
      colwisesums_thisblock[threadIdx.x +
                            y_in_thisblock * (blockDim.x + SHMEM_PAD_X)] =
          aggregated_sum;
      prev_aggregated_sum = aggregated_sum;
    }
    __syncthreads();
    blockwise_colwise_sums[global_tid_x + blockIdx.y * num_cols] =
        aggregated_sum;
  }

  if (global_tid_y < num_rows && threadIdx.x < blockDim.x) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
#pragma unroll
    for (int x_in_thisblock = 0; x_in_thisblock < blockDim.x;
         ++x_in_thisblock) {
      if (blockIdx.x * blockDim.x + x_in_thisblock < num_cols) {
        aggregated_sum =
            filter_coeff_0 *
            colwisesums_thisblock[x_in_thisblock +
                                  threadIdx.x * (blockDim.x + SHMEM_PAD_X)];
        aggregated_sum += filter_coeff_1 * prev_aggregated_sum;
      }
      prev_aggregated_sum = aggregated_sum;
    }
    blockwise_rowwise_sums[(blockIdx.y * blockDim.x + threadIdx.x) +
                           blockIdx.x * num_rows] = aggregated_sum;
    // Transposed to coalesce global memory access
  }
}

//__device__ inline int ilog2(int x) {
//	return INT_BIT_SIZE - __clz(x);
//}

__global__ void recursivefilter_step2_overblocksdown(
    int num_aggregated_rows, int num_cols, float filter_coeff_1_toblockdimypow,
    const float* __restrict__ blockwise_colwise_sums, float* __restrict__ aggregated_colwise_sums) {
  const int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_tid < num_cols) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
    for (int y_in_grid = 0; y_in_grid < num_aggregated_rows; ++y_in_grid) {
      aggregated_sum =
          __ldg((const float *)&blockwise_colwise_sums[global_tid +
                                                       y_in_grid * num_cols]) +
          filter_coeff_1_toblockdimypow * prev_aggregated_sum;
      prev_aggregated_sum = aggregated_sum;
      aggregated_colwise_sums[global_tid + y_in_grid * num_cols] =
          aggregated_sum;
    }
  }
}

__global__ void recursivefilter_step3_inoverblockscolsummedblocksright(
    int num_aggregated_rows, int num_cols, int num_aggregated_cols,
    int tableblockdim_x, float filter_coeff_0, float filter_coeff_1,
    const float* __restrict__ aggregated_colwise_sums,
    float* __restrict__ blockwise_rowwise_aggregatedcolsums) {
  const int global_tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_tid_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (global_tid_x < num_aggregated_cols &&
      global_tid_y < num_aggregated_rows) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
#pragma unroll
    for (int x_in_blockrow = 0; x_in_blockrow < tableblockdim_x;
         ++x_in_blockrow) {
      const int global_x_offset =
          global_tid_x * tableblockdim_x + x_in_blockrow;
      if (global_x_offset < num_cols) {
        aggregated_sum = filter_coeff_0 *
                         __ldg((const float *)&aggregated_colwise_sums
                                   [global_x_offset + global_tid_y * num_cols]);
        aggregated_sum += filter_coeff_1 * prev_aggregated_sum;
      }
      prev_aggregated_sum = aggregated_sum;
    }
    blockwise_rowwise_aggregatedcolsums[global_tid_y +
                                        global_tid_x * num_aggregated_rows] =
        aggregated_sum;
    // Transposed to coalesce global memory access
  }
}

__global__ void recursivefilter_step4_overblocksright(
    int num_rows, int num_aggregated_cols, int num_aggregated_rows,
	int num_rows_in2dblock,
    float filter_coeff_1, float filter_coeff_1_toblockdimxpow,
    const float* __restrict__ blockwise_rowwise_sums,
    const float* __restrict__ blockwise_rowwise_aggregatedcolsums,
    float* __restrict__ aggregated_rowwise_sums) {
  const int global_tid_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (global_tid_y < num_rows) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
	const int bwrwaggcs_row_id = global_tid_y / num_rows_in2dblock;
    for (int x_in_row = 0; x_in_row < num_aggregated_cols; ++x_in_row) {
      aggregated_sum =
		  __ldg((const float*)&blockwise_rowwise_sums[global_tid_y + x_in_row * num_rows]) +
          filter_coeff_1_toblockdimxpow * prev_aggregated_sum;
      // Transposed to coalesce global memory access
      if (bwrwaggcs_row_id > 0) {
        aggregated_sum +=
            powf(filter_coeff_1, (global_tid_y % num_rows_in2dblock) + 1) *
			__ldg((const float*)&blockwise_rowwise_aggregatedcolsums[(bwrwaggcs_row_id - 1) +
                                                x_in_row * num_aggregated_rows]);
        // Transposed to coalesce global memory access
      }
      prev_aggregated_sum = aggregated_sum;
      aggregated_rowwise_sums[global_tid_y + x_in_row * num_rows] =
          aggregated_sum;
      // Transposed to coalesce global memory access
    }
  }
}

__global__ void recursivefilter_step5_inblocksdownright(
    const float* __restrict__ input, int num_rows, int num_cols, float filter_coeff_0,
    float filter_coeff_1, const float* __restrict__ aggregated_colwise_sums,
    const float* __restrict__ aggregated_rowwise_sums, float* __restrict__ final_sums) {
  const int global_tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_tid_y = blockIdx.y * blockDim.x + threadIdx.x;
  // Yes, blockDim.x and threadIdx.x (not .y), as we have a 1D thread array
  // within a thread block
  const int thisblock_globalstart_x = blockIdx.x * blockDim.x;
  const int thisblock_globalstart_y = blockIdx.y * blockDim.x;
  extern __shared__ float aggregated_sums_thisblock[];
  
  const int y_in_thisblock_upper =
	  (thisblock_globalstart_y + blockDim.x >= num_rows)
	  ? (num_rows - thisblock_globalstart_y)
	  : blockDim.x;

  if (global_tid_x < num_cols) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
    if (blockIdx.y > 0) {
      prev_aggregated_sum =
		  __ldg((const float*)&aggregated_colwise_sums[global_tid_x + (blockIdx.y - 1) * num_cols]);
    }
    for (int y_in_thisblock = 0; y_in_thisblock < y_in_thisblock_upper;
         ++y_in_thisblock) {
      aggregated_sum =
          filter_coeff_0 *
		  __ldg((const float*)&input[global_tid_x +
                    (thisblock_globalstart_y + y_in_thisblock) * num_cols]) +
          filter_coeff_1 * prev_aggregated_sum;
      prev_aggregated_sum = aggregated_sum;
      aggregated_sums_thisblock[threadIdx.x +
                                        y_in_thisblock *
                                            (blockDim.x + SHMEM_PAD_X)] =
          aggregated_sum;
    }
    __syncthreads();
  }

  if (global_tid_y < num_rows) {
    float aggregated_sum, prev_aggregated_sum = 0.0f;
    if (blockIdx.x > 0) {
      prev_aggregated_sum =
		  __ldg((const float*)&aggregated_rowwise_sums[global_tid_y + (blockIdx.x - 1) * num_rows]);
      // Transposed to coalesce global memory access
    }
    const int x_in_thisblock_upper =
        (thisblock_globalstart_x + blockDim.x >= num_cols)
            ? (num_cols - thisblock_globalstart_x)
            : blockDim.x;
    for (int x_in_thisblock = 0; x_in_thisblock < x_in_thisblock_upper;
         ++x_in_thisblock) {
      aggregated_sum =
          filter_coeff_0 *
              aggregated_sums_thisblock[x_in_thisblock +
                                                threadIdx.x * (blockDim.x +
                                                               SHMEM_PAD_X)] +
          filter_coeff_1 * prev_aggregated_sum; // Yes, threadIdx.x (not .y)
      prev_aggregated_sum = aggregated_sum;
	  aggregated_sums_thisblock[x_in_thisblock + threadIdx.x * (blockDim.x + SHMEM_PAD_X)] = aggregated_sum;
    }
  }

  if (global_tid_x < num_cols) {
	  for (int y_in_thisblock = 0; y_in_thisblock < y_in_thisblock_upper; ++y_in_thisblock) {
		  final_sums[global_tid_x + (thisblock_globalstart_y + y_in_thisblock) * num_cols] = 
			  aggregated_sums_thisblock[threadIdx.x + y_in_thisblock * (blockDim.x + SHMEM_PAD_X)];
	  }
  }
}

float recursivefilter_downright_gpu(const CpuTable &input, float filter_coeff_0,
                                    float filter_coeff_1,
                                    config::BLOCK_SIZE_CLASS block_size_class,
                                    size_t num_kernel_runs,
                                    OUTPUT_STEP output_step,
                                    std::vector<CpuTable> &outputs) {
  if (input.num_rows() < 2) {
    throw std::runtime_error("Number of input rows must be at least 2");
  }
  if (input.num_cols() < 2) {
    throw std::runtime_error("Number of input cols must be at least 2");
  }
  if (num_kernel_runs < 1) {
    throw std::runtime_error("Number of kernel runs must be at least 1");
  }

  Logger::new_line("Input table dims: (" + std::to_string(input.num_cols()) +
                   ", " + std::to_string(input.num_rows()) + ")\n");

  chk_cu_err(hipInit(0));
  int device_count = -1;
  chk_cu_err(hipGetDeviceCount(&device_count));
  Logger::new_line("CUDA device count: " + std::to_string(device_count));
  hipDevice_t device;
  chk_cu_err(hipDeviceGet(&device, 0));
  hipCtx_t cudaContext;
  chk_cu_err(hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, device));

  if (input.num_rows() <= PRINT_LIMIT_Y && input.num_cols() <= PRINT_LIMIT_X) {
    Logger::new_line("\nInput:\n" + input.toString());
  }
  float *h_input =
      (float *)malloc(input.num_rows() * input.num_cols() * sizeof(float));
  for (int i_row = 0; i_row < input.num_rows(); ++i_row) {
    for (int i_col = 0; i_col < input.num_cols(); ++i_col) {
      h_input[i_col + i_row * input.num_cols()] = input.get(i_row, i_col);
    }
  }
  //texture<float, 2, hipReadModeElementType> input_texture;
  float *d_input;
  chk_cu_err(hipMalloc((void **)(&d_input),
                        input.num_rows() * input.num_cols() * sizeof(float)));
  chk_cu_err(hipMemcpy(d_input, h_input,
                        input.num_rows() * input.num_cols() * sizeof(float),
                        hipMemcpyHostToDevice));
 
  const size_t blockdim_2dgrid_x = get_blockdim_2dgrid_x(block_size_class);
  const size_t blockdim_2dgrid_y = get_blockdim_2dgrid_y(block_size_class);
  const size_t blockdim_1dgrid_x = get_blockdim_1dgrid_x(block_size_class);

  const dim3 blockdim_step1 =
      dim3(int(std::max(blockdim_2dgrid_x, blockdim_2dgrid_y)), 1, 1);
  // const dim3 threadblockdim_step2 = dim3(1, int(threadgriddim_localscale_y),
  // 1);
  const dim3 blockdim_step2 = dim3(int(blockdim_1dgrid_x), 1, 1);
  const dim3 blockdim_step3 = dim3(int(blockdim_1dgrid_x), 1, 1);
  const dim3 blockdim_step4 = dim3(1, int(blockdim_1dgrid_x), 1);
  const dim3 blockdim_step5 =
      dim3(int(std::max(blockdim_2dgrid_x, blockdim_2dgrid_y)), 1, 1);
  Logger::new_line("#threads in step 1's block: (" +
                   std::to_string(blockdim_step1.x) + ", " +
                   std::to_string(blockdim_step1.y) + ")");
  Logger::new_line("#threads in step 2's block: (" +
                   std::to_string(blockdim_step2.x) + ", " +
                   std::to_string(blockdim_step2.y) + ")");
  Logger::new_line("#threads in step 3's block: (" +
                   std::to_string(blockdim_step3.x) + ", " +
                   std::to_string(blockdim_step3.y) + ")");
  Logger::new_line("#threads in step 4's block: (" +
                   std::to_string(blockdim_step4.x) + ", " +
                   std::to_string(blockdim_step4.y) + ")");
  Logger::new_line("#threads in step 5's block: (" +
                   std::to_string(blockdim_step5.x) + ", " +
                   std::to_string(blockdim_step5.y) + ")");

  const size_t griddim_for2dblock_x =
      input.num_cols() % blockdim_2dgrid_x == 0
          ? input.num_cols() / blockdim_2dgrid_x
          : input.num_cols() / blockdim_2dgrid_x + 1;
  const size_t griddim_for2dblock_y =
      input.num_rows() % blockdim_2dgrid_y == 0
          ? input.num_rows() / blockdim_2dgrid_y
          : input.num_rows() / blockdim_2dgrid_y + 1;
  const size_t griddim_for1dblock_x =
      input.num_cols() % blockdim_1dgrid_x == 0
          ? input.num_cols() / blockdim_1dgrid_x
          : input.num_cols() / blockdim_1dgrid_x + 1;
  const size_t griddim_for1dblock_y =
	  input.num_rows() % blockdim_1dgrid_x == 0
	  ? input.num_rows() / blockdim_1dgrid_x
	  : input.num_rows() / blockdim_1dgrid_x + 1;
  
  const dim3 griddim_step1(int(griddim_for2dblock_x), int(griddim_for2dblock_y),
                           1);
  // const dim3 threadgriddim_step2(int(input.num_cols()), 1, 1);
  const dim3 griddim_step2(int(griddim_for1dblock_x), 1, 1);
  const dim3 griddim_step3(griddim_step1.x % blockdim_step3.x == 0
                               ? griddim_step1.x / blockdim_step3.x
                               : griddim_step1.x / blockdim_step3.x + 1,
                           int(griddim_for2dblock_y), 1);
  const dim3 griddim_step4(1, int(griddim_for1dblock_y), 1);
  const dim3 griddim_step5(int(griddim_for2dblock_x), int(griddim_for2dblock_y),
                           1);
  Logger::new_line("#blocks in step 1: (" + std::to_string(griddim_step1.x) +
                   ", " + std::to_string(griddim_step1.y) + ")");
  Logger::new_line("#blocks in step 2: (" + std::to_string(griddim_step2.x) +
                   ", " + std::to_string(griddim_step2.y) + ")");
  Logger::new_line("#blocks in step 3: (" + std::to_string(griddim_step3.x) +
                   ", " + std::to_string(griddim_step3.y) + ")");
  Logger::new_line("#blocks in step 4: (" + std::to_string(griddim_step4.x) +
                   ", " + std::to_string(griddim_step4.y) + ")");
  Logger::new_line("#blocks in step 5: (" + std::to_string(griddim_step5.x) +
                   ", " + std::to_string(griddim_step5.y) + ")");

  const size_t shmemsizebytes_step1 =
      (blockdim_step1.x + SHMEM_PAD_X) * blockdim_step1.x *
      sizeof(float); // Yes, .x as step1 has a 1D thread block spanning 2
                     // directions
  // const size_t shmemsizebytes_step2 = (tableblockdim_x + SHMEM_PAD_X) *
  //                                    threadgriddim_localscale_y *
  //                                    sizeof(float);
  const size_t shmemsizebytes_step5 =
      (blockdim_step5.x + SHMEM_PAD_X) * blockdim_step5.x *
      sizeof(float); // Yes, .x as step1 has a 1D thread block spanning 2
                     // directions

  const float filter_coeff_1_toblockdimypow =
      powf(filter_coeff_1, static_cast<float>(blockdim_2dgrid_y));
  const float filter_coeff_1_toblockdimxpow =
      powf(filter_coeff_1, static_cast<float>(blockdim_2dgrid_x));
  // const int n_step2_scansteps =
  // static_cast<int>(log2(threadblockdim_step2.y));

  float *d_step1_blockwise_colwise_sums;
  const size_t n_step1_inblocksdown_rows = griddim_step1.y;
  chk_cu_err(
      hipMalloc((void **)(&d_step1_blockwise_colwise_sums),
                 input.num_cols() * n_step1_inblocksdown_rows * sizeof(float)));
  Logger::new_line("Blockwise-colwise table dims: (" +
                   std::to_string(input.num_cols()) + ", " +
                   std::to_string(n_step1_inblocksdown_rows) + ")");

  float
      *d_step1_blockwise_rowwise_sums; // Transposed to coalesce global memory accesses
  const size_t n_step1_inblocksdownright_cols = griddim_step1.x;
  chk_cu_err(hipMalloc((void **)(&d_step1_blockwise_rowwise_sums),
                        input.num_rows() * n_step1_inblocksdownright_cols *
                            sizeof(float)));
  Logger::new_line("Blockwise-rowwise table dims (transposed!): (" +
                   std::to_string(input.num_rows()) + ", " +
                   std::to_string(n_step1_inblocksdownright_cols) + ")");

  float *d_step2_aggregated_colwise_sums;
  const size_t n_step2_overblocksdown_rows = n_step1_inblocksdown_rows;
  chk_cu_err(hipMalloc((void **)(&d_step2_aggregated_colwise_sums),
                        input.num_cols() * n_step2_overblocksdown_rows *
                            sizeof(float)));
  Logger::new_line("Aggregated colwise table dims: (" +
                   std::to_string(input.num_cols()) + ", " +
                   std::to_string(n_step2_overblocksdown_rows) + ")");

  float *d_step3_blockwise_rowwise_aggregatedcolsums;	// Transposed to coalesce global memory accesses
  const size_t n_step3_inoverblockscolsummedblocksright_rows =
      n_step2_overblocksdown_rows;
  const size_t n_step3_inoverblockscolsummedblocksright_cols =
      n_step1_inblocksdownright_cols;
  chk_cu_err(hipMalloc((void **)(&d_step3_blockwise_rowwise_aggregatedcolsums),
                        n_step3_inoverblockscolsummedblocksright_cols *
                            n_step3_inoverblockscolsummedblocksright_rows *
                            sizeof(float)));
  Logger::new_line(
      "Blockwise-rowwise aggregatedcolsum table dims: (" +
      std::to_string(n_step3_inoverblockscolsummedblocksright_cols) + ", " +
      std::to_string(n_step3_inoverblockscolsummedblocksright_rows) + ")");

  float *d_step4_aggregated_rowwise_sums;	// Transposed to coalesce global memory accesses
  const size_t n_recursivefilter_step4_overblocksright_cols =
      n_step1_inblocksdownright_cols;
  chk_cu_err(hipMalloc((void **)(&d_step4_aggregated_rowwise_sums),
                        n_recursivefilter_step4_overblocksright_cols *
                            input.num_rows() * sizeof(float)));
  Logger::new_line(
      "Aggregated rowwise sum table dims: (" +
      std::to_string(n_recursivefilter_step4_overblocksright_cols) + ", " +
      std::to_string(input.num_rows()) + ")");

  float *d_step5_finalsums_rowmajor;
  chk_cu_err(hipMalloc((void **)(&d_step5_finalsums_rowmajor), input.num_rows() * input.num_cols() * sizeof(float)));

  float run_time_allruns_ms = -1.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for (size_t i_run = 0; i_run < num_kernel_runs; ++i_run) {
    recursivefilter_step1_inblocksdownright<<<griddim_step1, blockdim_step1,
                                              shmemsizebytes_step1>>>(
        d_input, int(input.num_rows()), int(input.num_cols()), filter_coeff_0,
        filter_coeff_1, d_step1_blockwise_colwise_sums, d_step1_blockwise_rowwise_sums);
    recursivefilter_step2_overblocksdown<<<griddim_step2, blockdim_step2>>>(
        int(n_step2_overblocksdown_rows), int(input.num_cols()),
        filter_coeff_1_toblockdimypow, d_step1_blockwise_colwise_sums,
        d_step2_aggregated_colwise_sums);
    recursivefilter_step3_inoverblockscolsummedblocksright<<<griddim_step3,
                                                             blockdim_step3>>>(
        int(n_step3_inoverblockscolsummedblocksright_rows),
        int(input.num_cols()),
        int(n_step3_inoverblockscolsummedblocksright_cols),
        int(blockdim_2dgrid_x), filter_coeff_0, filter_coeff_1,
        d_step2_aggregated_colwise_sums, d_step3_blockwise_rowwise_aggregatedcolsums);
    recursivefilter_step4_overblocksright<<<griddim_step4, blockdim_step4>>>(
        int(input.num_rows()),
        int(n_recursivefilter_step4_overblocksright_cols),
        int(n_step3_inoverblockscolsummedblocksright_rows), blockdim_step1.x, filter_coeff_1,
        filter_coeff_1_toblockdimxpow, d_step1_blockwise_rowwise_sums,
        d_step3_blockwise_rowwise_aggregatedcolsums, d_step4_aggregated_rowwise_sums);
    recursivefilter_step5_inblocksdownright<<<griddim_step5, blockdim_step5,
                                              shmemsizebytes_step5>>>(
        d_input, int(input.num_rows()), int(input.num_cols()), filter_coeff_0,
        filter_coeff_1, d_step2_aggregated_colwise_sums, d_step4_aggregated_rowwise_sums,
        d_step5_finalsums_rowmajor);
  }
  // hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&run_time_allruns_ms, start, stop);
  const float run_time_1run_ms = run_time_allruns_ms / float(num_kernel_runs);
  Logger::new_line(
      "\nKernel execution time for " + std::to_string(input.num_cols()) + "x" +
      std::to_string(input.num_rows()) +
      " [ms]: " + std::to_string(run_time_1run_ms) + " (average of " +
      std::to_string(num_kernel_runs) + " runs)");

  float *h_blockwise_colwise_sums = (float *)malloc(
      n_step1_inblocksdown_rows * input.num_cols() * sizeof(float));
  chk_cu_err(
      hipMemcpy(h_blockwise_colwise_sums, d_step1_blockwise_colwise_sums,
                 n_step1_inblocksdown_rows * input.num_cols() * sizeof(float),
                 hipMemcpyDeviceToHost));
  CpuTable blockwise_colwise_sums(n_step1_inblocksdown_rows, input.num_cols(),
                                  h_blockwise_colwise_sums);
  if (n_step1_inblocksdown_rows <= 12 && input.num_cols() <= 12) {
    Logger::new_line("\nBlockwise-colwise table (light blue):\n" +
                     blockwise_colwise_sums.toString());
  }
  float *h_step1_inblocksdownright = (float *)malloc(
      n_step1_inblocksdownright_cols * input.num_rows() * sizeof(float));
  chk_cu_err(hipMemcpy(h_step1_inblocksdownright, d_step1_blockwise_rowwise_sums,
                        n_step1_inblocksdownright_cols * input.num_rows() *
                            sizeof(float),
                        hipMemcpyDeviceToHost));
  CpuTable blockwise_rowwise_sums(n_step1_inblocksdownright_cols,
                                  input.num_rows(), h_step1_inblocksdownright);

  blockwise_rowwise_sums.transpose();
  if (input.num_rows() <= 12 && n_step1_inblocksdownright_cols <= 12) {
    Logger::new_line("\nBlockwise-rowwise table (light green):\n" +
                     blockwise_rowwise_sums.toString());
  }

  float *h_step2_overblocksdown = (float *)malloc(
      n_step2_overblocksdown_rows * input.num_cols() * sizeof(float));
  chk_cu_err(
      hipMemcpy(h_step2_overblocksdown, d_step2_aggregated_colwise_sums,
                 n_step2_overblocksdown_rows * input.num_cols() * sizeof(float),
                 hipMemcpyDeviceToHost));
  CpuTable aggregated_blockwise_colwise_sums(
      n_step2_overblocksdown_rows, input.num_cols(), h_step2_overblocksdown);
  if (n_step2_overblocksdown_rows <= 12 && input.num_cols() <= 12) {
    Logger::new_line("\nAggregated blockwise-colwise table (dark blue):\n" +
                     aggregated_blockwise_colwise_sums.toString());
  }

  float *h_step3_inoverblockscolsummedblocksright = (float *)malloc(
      n_step3_inoverblockscolsummedblocksright_cols *
      n_step3_inoverblockscolsummedblocksright_rows * sizeof(float));
  chk_cu_err(hipMemcpy(h_step3_inoverblockscolsummedblocksright,
                        d_step3_blockwise_rowwise_aggregatedcolsums,
                        n_step3_inoverblockscolsummedblocksright_cols *
                            n_step3_inoverblockscolsummedblocksright_rows *
                            sizeof(float),
                        hipMemcpyDeviceToHost));
  CpuTable blockwise_rowwise_aggregatedcolsums(
      n_step3_inoverblockscolsummedblocksright_cols,
      n_step3_inoverblockscolsummedblocksright_rows,
      h_step3_inoverblockscolsummedblocksright);
  blockwise_rowwise_aggregatedcolsums.transpose();
  if (n_step3_inoverblockscolsummedblocksright_rows <= 12 &&
      n_step3_inoverblockscolsummedblocksright_cols <= 12) {
    Logger::new_line("\nBlockwise-rowwise aggregatedcolsum table (red):\n" +
                     blockwise_rowwise_aggregatedcolsums.toString());
  }

  float *h_step4_overblocksright =
      (float *)malloc(n_recursivefilter_step4_overblocksright_cols *
                      input.num_rows() * sizeof(float));
  chk_cu_err(hipMemcpy(h_step4_overblocksright, d_step4_aggregated_rowwise_sums,
                        n_recursivefilter_step4_overblocksright_cols *
                            input.num_rows() * sizeof(float),
                        hipMemcpyDeviceToHost));
  CpuTable aggregated_rowwise_sums(n_recursivefilter_step4_overblocksright_cols,
                                   input.num_rows(), h_step4_overblocksright);
  aggregated_rowwise_sums.transpose();
  if (input.num_rows() <= 12 &&
      n_recursivefilter_step4_overblocksright_cols <= 12) {
    Logger::new_line("\nAggregated rowwise table (dark green):\n" +
                     aggregated_rowwise_sums.toString());
  }

  float *h_step5_finalsums_rowmajor =
      (float *)malloc(input.num_rows() * input.num_cols() * sizeof(float));
  chk_cu_err(hipMemcpy(h_step5_finalsums_rowmajor, d_step5_finalsums_rowmajor,
                        input.num_rows() * input.num_cols() * sizeof(float),
                        hipMemcpyDeviceToHost));
  CpuTable finalsums_rowmajor(input.num_rows(), input.num_cols(), h_step5_finalsums_rowmajor);
  if (finalsums_rowmajor.num_rows() <= PRINT_LIMIT_Y &&
	  finalsums_rowmajor.num_cols() <= PRINT_LIMIT_X) {
	  Logger::new_line("\nFinal sums:\n" + finalsums_rowmajor.toString());
  }

  if (SAVE_TABLES_TO_CSV) {
	  blockwise_colwise_sums.saveToCsv("step1_blockwise_colwise_sums.csv");
	  blockwise_rowwise_sums.saveToCsv("step1_blockwise_rowwise_sums.csv");
	  aggregated_blockwise_colwise_sums.saveToCsv(
		  "step2_aggregated_blockwise_colwise_sums.csv");
	  blockwise_rowwise_aggregatedcolsums.saveToCsv(
		  "step2_blockwise_rowwise_aggregatedcolsums.csv");
	  aggregated_rowwise_sums.saveToCsv("step4_aggregated_rowwise_sums.csv");
	  finalsums_rowmajor.saveToCsv("step5_finalsums_rowmajor.csv");
  }

  switch (output_step) {
  case STEP_1: {
    outputs[0].reset(n_step1_inblocksdown_rows, input.num_cols(),
                     h_blockwise_colwise_sums);
    outputs[1].reset(n_step1_inblocksdownright_cols, input.num_rows(),
                     h_step1_inblocksdownright);
    outputs[1].transpose();
    break;
  }
  case STEP_2: {
    outputs[0].reset(n_step2_overblocksdown_rows, input.num_cols(),
                     h_step2_overblocksdown);
    break;
  }
  case STEP_3: {
    outputs[0].reset(n_step3_inoverblockscolsummedblocksright_cols,
                     n_step3_inoverblockscolsummedblocksright_rows,
                     h_step3_inoverblockscolsummedblocksright);
    outputs[0].transpose();
    break;
  }
  case STEP_4: {
    outputs[0].reset(n_recursivefilter_step4_overblocksright_cols,
                     input.num_rows(), h_step4_overblocksright);
    outputs[0].transpose();
    break;
  }
  case STEP_5: {
	outputs[0].reset(input.num_rows(), input.num_cols(), h_step5_finalsums_rowmajor);
    break;
  }
  default:
	  throw std::runtime_error("Invalid output step requested: " + std::to_string(output_step));
  }

  chk_cu_err(hipFree(d_input));
  free(h_input);
  chk_cu_err(hipFree(d_step1_blockwise_colwise_sums));
  free(h_blockwise_colwise_sums);
  chk_cu_err(hipFree(d_step1_blockwise_rowwise_sums));
  free(h_step1_inblocksdownright);
  chk_cu_err(hipFree(d_step2_aggregated_colwise_sums));
  free(h_step2_overblocksdown);
  chk_cu_err(hipFree(d_step3_blockwise_rowwise_aggregatedcolsums));
  free(h_step3_inoverblockscolsummedblocksright);
  chk_cu_err(hipFree(d_step4_aggregated_rowwise_sums));
  free(h_step4_overblocksright);
  chk_cu_err(hipFree(d_step5_finalsums_rowmajor));
  free(h_step5_finalsums_rowmajor);

  return run_time_1run_ms;
}

} // namespace gpuacademy
